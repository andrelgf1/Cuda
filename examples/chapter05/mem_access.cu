
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define ROWS 999
#define COLUMNS 999
int m[ROWS][COLUMNS];
//Taking both dimensions same so that while running the loops,
//number of operations (comparisons, iterations, initializations)
//are exactly the same. Refer this for more
// https://www.geeksforgeeks.org/a-nested-loop-puzzle/

int main() {
	int i, j;
	clock_t start, stop;
	double d = 0.0;

	int count = 1;

	start = clock();
	for (i = 0; i < ROWS; i++)
		for (j = 0; j < COLUMNS; j++)
			m[i][j] = count++;

	stop = clock();
	d = (double)(stop - start) / CLOCKS_PER_SEC;
	printf("The run-time of %d x %d matrix with row major order is %lf\n", ROWS, COLUMNS, d);

	count = 1;
	start = clock();
	for (j = 0; j < COLUMNS; j++)
		for (i = 0; i < ROWS; i++)
			m[i][j] = count++;

	stop = clock();
	d = (double)(stop - start) / CLOCKS_PER_SEC;
	printf("The run-time of %d x %d matrix with column major order is %lf\n\n", ROWS, COLUMNS, d);
}
