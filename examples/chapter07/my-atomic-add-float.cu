#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>

/**
 * This example illustrates implementation of custom atomic operations using
 * CUDA's built-in atomicCAS function to implement atomic signed 32-bit integer
 * addition.
 **/

__device__ float myAtomicAddFloat(float *address, float incr) {

    // Convert address to point to a supported type of the same size
    unsigned int *typedAddress = (unsigned int *)address;
    // Stored the expected and desired float values as an unsigned int
    float currentVal = *address;
    unsigned int expected = __float2uint_rn(currentVal);
    unsigned int desired = __float2uint_rn(currentVal + incr);
    int oldIntValue = atomicCAS(typedAddress, expected, desired);

    while (oldIntValue != expected) {
        expected = oldIntValue;
        /*
        * Convert the value read from typedAddress to a float, increment,
        * and then convert back to an unsigned int
        */
        desired = __float2uint_rn(__uint2float_rn(oldIntValue) + incr);
        oldIntValue = atomicCAS(typedAddress, expected, desired);
    }

    return __uint2float_rn(oldIntValue);
}


__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}

// __global__ void kernel(int *sharedInteger)
__global__ void kernel(float *sharedFloat)
{
    // myAtomicAdd(sharedInteger, 1);
    myAtomicAddFloat(sharedFloat, 1.0f);
}

int main(int argc, char **argv)
{
    // int h_sharedInteger;
    // int *d_sharedInteger;
    float h_sharedFloat;
    float *d_sharedFloat;

    CHECK(hipMalloc((void **)&d_sharedFloat, sizeof(float)));
    CHECK(hipMemset(d_sharedFloat, 0x00, sizeof(float)));

    kernel<<<4, 128>>>(d_sharedFloat);

    CHECK(hipMemcpy(&h_sharedFloat, d_sharedFloat, sizeof(float),
                     hipMemcpyDeviceToHost));
    // printf("4 x 128 increments led to value of %f\n", h_sharedFloat);
    printf("4 x 128 increments led to value of %u\n", *(unsigned int*) &h_sharedFloat);

    return 0;
}

