#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;

__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    // alter the value
    devData += 2.0f;
}

int main(void)
{
    // initialize the global variable
    float value = 3.14f;
    float *d_add_p;
    // CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    CHECK(hipGetSymbolAddress((void **)&d_add_p, devData));
    CHECK(hipMemcpy(d_add_p, &value, sizeof(float), hipMemcpyHostToDevice));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // copy the global variable back to the host
    // CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    CHECK(hipMemcpy(&value, d_add_p, sizeof(float), hipMemcpyDeviceToHost));
    printf("Host:   the value changed by the kernel to %f\n", value);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
